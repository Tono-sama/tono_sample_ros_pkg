#include "hip/hip_runtime.h"
#include "include.hpp"

__global__ void cuda_helloworld(){
    printf("hello, world from CUDA !!! (thredIdx=%d)\n", threadIdx.x);
}

void call_cuda_helloworld(){
    cuda_helloworld<<<1,1>>>();
    hipDeviceSynchronize();
}